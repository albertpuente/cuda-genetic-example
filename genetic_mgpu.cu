/*
Compile:
    
    make 

Usage (3D viewer):

    ./genetic > data && ./geneticViewer data
    
Usage (debug):

    ./genetic

Jan Mas Rovira
Andrés Mingorance López
Albert Puente Encinas
*/

#include <stdio.h>  // e.g. printf
#include <stdlib.h> // e.g. malloc, RAND_MAX, exit
#include <math.h>   // e.g. sin, abs
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define CURAND hiprand_uniform(&localState)

#define MULTI_GPU true
#define DUMP true

// Genetic algorithm parameters
#define N 1024*2
#define N_POINTS 256*2
#define ITERATION_LIMIT 1200
#define GOAL_SCORE -1.0
#define POINT_SET_MUTATION_PROB 0.5
#define POINT_MUTATION_PROB 0.25
#define N_SURVIVORS N/4
#define POINT_RADIUS 0.1
#define OBSTACLE_RADIUS 2.0
#define MAX_DELTA 2
#define MAX_TRIES 1e4   // max amount of times we tries to find a position for a point

// Obstacles
#define CHECK_OBSTACLES true
#define CHECK_COLLISIONS false

// Deterministic algorithm (testing purposes)
#define SEED 27
#define RAND01 ((float)rand()/(float)(RAND_MAX))

// c++ style
#define bool int
#define true 1
#define false 0

// MultiGPU
#define ForEachGPU for (int GPU_X = 0; GPU_X < nGPUs; ++GPU_X)
unsigned int bestGPU;
int* seeds;
hiprandState** devStates;

// Timers
unsigned long long mutationTime;
unsigned long long reproductionTime;
unsigned long long sortingTime;
unsigned long long evaluationTime;
unsigned long long initialGenTime;
unsigned long long totalTime;

inline void tic(unsigned long long* time) {
    struct timeval t;
    gettimeofday(&t, NULL);
    *time = t.tv_sec*1000000 + t.tv_usec - *time;
}
#define toc tic
//inline void toc(unsigned long long* time) { tic(time); }

typedef struct {
    float x, y, z; // Position
} Point;

typedef struct {
    Point points[N_POINTS];
    float score;
} PointSet;

typedef struct {
    PointSet pointSets[N];
    float maxScore;
} Population;

typedef struct {
    Point centre;
    float radius;
} Obstacle;

#define N_OBSTACLES 27
Obstacle obstacles[N_OBSTACLES];
Point destination;

// CUDA Variables
unsigned int nThreads = 1024;
unsigned int nBlocks = N/nThreads;  // N multiple de nThreads
int nGPUs;
// GPU Pointers
Obstacle** gpu_obstacles;
Point** gpu_destination;

__device__ inline bool cuda_randomChoice(float probability, hiprandState* localState) {
    if (hiprand_uniform(localState) <= probability) return true;
    else return false;    
}

void checkCudaError(char msg[]) {
    hipError_t error;
    error = hipGetLastError();
    if (error) {
        printf("Error: %s: %s\n", msg, hipGetErrorString(error));
        exit(1);
    }
}

__device__ inline float cuda_squared_dist(Point* a, Point* b) {
    return (float) (a->x-b->x)*(a->x-b->x)+(a->y-b->y)*(a->y-b->y)+(a->z-b->z)*(a->z-b->z);
}

// check if point collides with any of the obstacles
__device__ bool collidesWithObstacles(Point* p, Obstacle* obstacles) {
    if (!CHECK_OBSTACLES) return false;
    for (int i = 0; i < N_OBSTACLES; ++i) {
        Obstacle o = obstacles[i];
        //                                    mult. by itself since we compare to squared dist
        if (cuda_squared_dist(p, &o.centre) < (POINT_RADIUS + o.radius)*(POINT_RADIUS + o.radius)) {
            return true;
        }
    }
    return false;
}

__device__ bool cuda_collides(Point* p, PointSet* PS, int from, int to, Obstacle* obstacles) {
    float squared_d = (POINT_RADIUS*POINT_RADIUS)*4;
    if (CHECK_COLLISIONS)
        for (int i = from; i < to; ++i) {
            if (cuda_squared_dist(p, &PS->points[i]) < squared_d) {
                return true;
            }
        }
    if (CHECK_OBSTACLES)
        for (int i = 0; i < N_OBSTACLES; ++i) {
            Obstacle o = obstacles[i];
            if (cuda_squared_dist(p, &o.centre) < (POINT_RADIUS + o.radius)*(POINT_RADIUS + o.radius)) {
                return true;
            }
        }
    return false;
}

__global__ void kernel_generateInitialPopulation(Population* P, 
                    Obstacle* obstacles, int* idxs, hiprandState* state) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Indexs initialization
    idxs[id] = id;
    
    hiprandState localState = state[id];
    
    float range = POINT_RADIUS * pow((float)N_POINTS, 1.0f/3.0f) * 10;    
    
    /*
    float r1 = curand_uniform(&localState)/(float)(RAND_MAX);
    float r2 = curand_uniform(&localState);///(float)(RAND_MAX);
    float r3 = curand_uniform(&localState);///(float)(RAND_MAX);
    
    printf("%f %f %f\n", r1, r2, r3);
    */
    
    for (int j = 0; j < N_POINTS; ++j) {
        PointSet* PS = &(P->pointSets[id]);
        Point* p = &(PS->points[j]); // p is passed to 'collides' via PS
        p->x = CURAND * range + 12.5;
        p->y = CURAND * range + 12.5;
        p->z = CURAND * range + 12.5;
        
        int tries = 0;
        while (tries < MAX_TRIES && cuda_collides(p, PS, 0, j, obstacles)) {
            p->x = CURAND * range + 12.5;
            p->y = CURAND * range + 12.5;
            p->z = CURAND * 5.0 + 12.5;
            ++tries;
        }
        if (tries == MAX_TRIES) {
            printf("Error during the generation of the initial population, MAX TRIES REACHED\n");
            //exit(1);
        }
    }
}

__global__ void setup_kernel(hiprandState *state, int seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init (seed, id, 0, &state[id] );
}

void generateInitialPopulation(Population** gpu_P, int** gpu_idxs) {
    tic(&initialGenTime);
       
    ForEachGPU {
        hipSetDevice(GPU_X);
        // kernel 
        kernel_generateInitialPopulation<<<nBlocks, nThreads>>>(gpu_P[GPU_X],
                                                                gpu_obstacles[GPU_X], 
                                                                gpu_idxs[GPU_X],
                                                                devStates[GPU_X]);
        checkCudaError((char *) "kernel call in generateInitialPopulation\n");
    }
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }

    toc(&initialGenTime);
}

__device__ inline float heur_1(Point* P) {
    return fabs(P->y - 3.0*sin(P->x/2.0)) + fabs(P->z - 3.0*cos(P->x/2.0));
}


__device__ inline float heur_2(Point* P, Point* destination) {
    return cuda_squared_dist(P, destination);
}

__global__ void kernel_evaluate(Population* P, Point* destination) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    PointSet* C = &P->pointSets[id];
    C->score = 0;
    for (int j = 0; j < N_POINTS; j++) {
        Point* E = &C->points[j];
        C->score += heur_2(E, destination);
    }
}

void evaluate(Population** gpu_P) {
    tic(&evaluationTime);
    
    // kernel
    ForEachGPU {
        hipSetDevice(GPU_X);
        kernel_evaluate<<<nBlocks, nThreads>>>(gpu_P[GPU_X], gpu_destination[GPU_X]);
        checkCudaError((char *) "kernel call in evaluate");
    }
    
    // wait
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }
    
    toc(&evaluationTime);
}

//////////////////////////////////////////
//////////////  CUDA QUICK SORT
//////         

#define MAX_DEPTH       16
#define INSERTION_SORT  32

// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
__device__ void selection_sort(Population* P, int* idxs, int left, int right ) {
    for (int i = left ; i <= right ; ++i) {
        float min_score = P->pointSets[ idxs[i] ].score;
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i + 1 ; j <= right ; ++j) {
            float score_j = P->pointSets[ idxs[j] ].score;
            if (score_j < min_score) {
                min_idx = j;
                min_score = score_j;
            }
        }

        // Swap the values.
        if (i != min_idx) {
            int aux = idxs[i];
            idxs[i] = idxs[min_idx];
            idxs[min_idx] = aux;
        }
    }
}

__global__ void dynamic_quicksort(Population* P, int* idxs, int left, int right, int depth) {
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT) {
        selection_sort(P, idxs, left, right);
        return;
    }
    
    int lindex = left;
    int rindex = right;
    float pscore = P->pointSets[ idxs[(left+right)/2] ].score; // Pivot

    // Do the partitioning.
    while (lindex <= rindex) {
        // Find the next left- and right-hand values to swap
        float lscore = P->pointSets[ idxs[lindex] ].score; 
        float rscore = P->pointSets[ idxs[rindex] ].score;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lscore < pscore) {
            lindex++;
            lscore = P->pointSets[ idxs[lindex] ].score; 
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rscore > pscore) {
            rindex--;
            rscore = P->pointSets[ idxs[rindex] ].score;
        }

        // If the swap points are valid, do the swap!
        if (lindex <= rindex) {
            
            int aux = idxs[lindex];
            idxs[lindex] = idxs[rindex];
            idxs[rindex] = aux;
            
            lindex++;
            rindex--;
        }
    }

    // Now the recursive part
    // Launch a new block to sort the left part.
    if (left < rindex) {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        dynamic_quicksort<<< 1, 1, 0, s >>>(P, idxs, left, rindex, depth + 1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if (lindex < right) {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        dynamic_quicksort<<< 1, 1, 0, s1 >>>(P, idxs, lindex, right, depth + 1);
        hipStreamDestroy(s1);
    }
}

__global__ void copyBestPointSet(Population* P, int* idxs, PointSet* best) {
    *best = P->pointSets[ idxs[0] ];
}

__global__ void checkSort(Population* P, int* idxs) {
    for (int i = 1; i < N; ++i) {
       float a = P->pointSets[ idxs[i - 1] ].score;
       float b = P->pointSets[ idxs[i] ].score;
       if (a > b) printf("SORT IS NOT WORKING\n");
    }
}


void sort(Population** gpu_P, int** gpu_idxs, PointSet** best) {
    tic(&sortingTime);
    
    ForEachGPU {
        hipSetDevice(GPU_X);        
        dynamic_quicksort<<<1, 1>>>(gpu_P[GPU_X], gpu_idxs[GPU_X], 0, N-1, 0);
        checkCudaError((char *) "kernel call in sort");
    }
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }
   
    
    PointSet** gpu_best = (PointSet**) malloc(nGPUs*sizeof(PointSet*));
    
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipMalloc(&gpu_best[GPU_X], sizeof(PointSet));
        checkCudaError((char *) "hipMalloc bestPointSet");
    
        copyBestPointSet<<<1, 1>>>(gpu_P[GPU_X], gpu_idxs[GPU_X], gpu_best[GPU_X]);
        checkCudaError((char *) "kernel copy best pointset");
    }
    
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }
    
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipMemcpy(best[GPU_X], gpu_best[GPU_X], sizeof(PointSet), hipMemcpyDeviceToHost);
        checkCudaError((char *) "copy of best point set");
    }
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }
    
    bestGPU = 0;
    if (!DUMP) printf("GPU_0 score: %f\n", best[0]->score);
    for (int i = 1; i < nGPUs; ++i) {
        if (!DUMP) printf("GPU_%i score: %f\n", i, best[i]->score);
        if (best[i]->score < best[bestGPU]->score) {
            bestGPU = i;
        }
    }
    
    toc(&sortingTime);
}

__device__ void mix(PointSet* AP, PointSet* AQ, Obstacle* obstacles, 
                                                    hiprandState* localState) {

    for (int i = 0; i < N_POINTS; ++i) {
        
        if (!cuda_randomChoice(POINT_MUTATION_PROB, localState)) {
            AQ->points[i] = AP->points[i];
            continue;
        }           
   
        int tries = 0;
        Point p;
        while (tries < MAX_TRIES) {            
            // Choose a reference point
            int j = hiprand_uniform(localState)*(N_POINTS-1);
            
            // Calculate the direction from AP[i] to AP[j]
            float dx =  AP->points[j].x - AP->points[i].x;
            float dy =  AP->points[j].y - AP->points[i].y;
            float dz =  AP->points[j].z - AP->points[i].z;
            // "Normalization" ||direction|| = 0.5
            float norm = sqrt(pow(dx,2)+pow(dy,2)+pow(dz,2));
            norm *= (1.0/MAX_DELTA);
            norm /= hiprand_uniform(localState); // move a random portion of MAX_DELTA
            if (norm < 1e-4 && norm > -1e-4) {
                dx = 0;
                dy = 0;
                dz = 0;
            }
            else {
                dx /= norm;
                dy /= norm;  
                dz /= norm;   
            }       
            
            // 50% of getting closer, 50% of getting further away from the ref point
            if (cuda_randomChoice(0.5f, localState)) {
                p.x = AP->points[i].x + dx;
                p.y = AP->points[i].y + dy;
                p.z = AP->points[i].z + dz;
            }
            else {
                p.x = AP->points[i].x - dx;
                p.y = AP->points[i].y - dy;
                p.z = AP->points[i].z - dz;
            }
            // if the point doesn't collide with a point that has already moved
            if (!cuda_collides(&p, AQ, 0, i, obstacles) &&
                // and it doesn't collide with a point that has yet to be moved
                // (this 2nd check prevents inconsistencies like a point being unable to move at all)
                !cuda_collides(&p, AP, i + 1, N_POINTS, obstacles))
                    break;
            ++tries;
        }
        if (tries == MAX_TRIES) {
            //printf("Error during the mix() of points\n");
            //exit(1);
            p = AP->points[i];
        }
        AQ->points[i] = p;
    }
}

__device__ void randomMove(PointSet* AP, PointSet* AQ, Obstacle* obstacles, 
                                                hiprandState* localState) {
    for (int i = 0; i < N_POINTS; ++i) {
        
        if (!cuda_randomChoice(POINT_MUTATION_PROB, localState)) {
            AQ->points[i] = AP->points[i];
            continue;
        }
        int tries = 0;
        Point p;
        while (tries < MAX_TRIES) {
            p.x = AP->points[i].x + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
            p.y = AP->points[i].y + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
            p.z = AP->points[i].z + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
            // if the point doesn't collide with a point that has already moved
            if (!cuda_collides(&p, AQ, 0, i, obstacles) &&
                // and it doesn't collide with a point that has yet to be moved
                // (this 2nd check prevents inconsistencies like a point being unable to move at all)
                !cuda_collides(&p, AP, i + 1, N_POINTS, obstacles))
                    break;
            ++tries;
        }
        if (tries == MAX_TRIES) {
            //printf("Error during the mix() of points\n");
            //exit(1);
            p = AP->points[i];
        }

        AQ->points[i] = p;
    } 
}

__global__ void kernel_mutate(Population* P, Population* Q, Obstacle* obstacles, 
                                                             hiprandState* state) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    hiprandState localState = state[id];
    
    PointSet* AP = &P->pointSets[id];    // original points
    PointSet* AQ = &Q->pointSets[id];    // mutated points    
    
    if (cuda_randomChoice(POINT_SET_MUTATION_PROB, &localState)) { // Mutate
        if (cuda_randomChoice(0.5f, &localState)) {
            mix(AP, AQ, obstacles, &localState);
        }
        else {
            randomMove(AP, AQ, obstacles, &localState);
        }
    }        
    else { // Copy
        *AQ = *AP;
    }
}


__device__ void mixFlat(Point* P, PointSet* AP, Point* Q, Obstacle* obstacles, hiprandState* localState) {

    Point p;
    do {            
        // Choose a reference point
        int j = hiprand_uniform(localState)*(N_POINTS-1);
        
        // Calculate the direction from AP[i] to AP[j]
        float dx =  P->x - AP->points[j].x;
        float dy =  P->y - AP->points[j].y;
        float dz =  P->z - AP->points[j].z;
        // pseudonormalization
        float norm = sqrt(dx*dx + dy*dy + dz*dz);
        norm *= 1.0/(MAX_DELTA);
        norm /= hiprand_uniform(localState); // move a random portion of MAX_DELTA
        float val = 1.0 / norm;
        if (norm < 1e-4 && norm > -1e-4) val = 0;
        dx *= val;
        dy *= val;  
        dz *= val;
            
        // 50% of getting closer, 50% of getting further away from the ref point
        int closer = 1;
        if (cuda_randomChoice(0.5f, localState)) closer = -1;
        p.x = P->x + dx*closer;
        p.y = P->y + dy*closer;
        p.z = P->z + dz*closer;
        // if the point doesn't collide with a point that has already moved
    } while (collidesWithObstacles(&p, obstacles));
    
    *Q = p;
}

__device__ void randomMoveFlat(Point* P, Point* Q, Obstacle* obstacles, hiprandState* localState) {
    Point p;
    p.x = P->x + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
    p.y = P->y + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
    p.z = P->z + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
    // if the point doesn't collide with a point that has already moved
    while (collidesWithObstacles(&p, obstacles)) {
        p.x = P->x + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
        p.y = P->y + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
        p.z = P->z + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
    }
    *Q = *P;
}

__global__ void kernel_mutateFlat(Population* P, Population* Q, Obstacle* obstacles, hiprandState* state) {    
    
    int ps_id = blockIdx.x * blockDim.x + threadIdx.x;
    int point_id = blockIdx.y * blockDim.y + threadIdx.y;
    
    // decide, for each pointset, its mutation type
    // 0 = no mutation, 1 = mix, 2 = randomMove
    __shared__ int mutate_pointset[N]; // store if pointsets need to be mutated
    if (point_id == 0) {
        if (cuda_randomChoice(POINT_SET_MUTATION_PROB, &state[ps_id])) {
            if (cuda_randomChoice(0.5, &state[ps_id])) mutate_pointset[ps_id] = 1;
            else mutate_pointset[ps_id] = 2;
        }
        else mutate_pointset[ps_id] = 0;
    }
    __syncthreads();
    
    // now treat points    
    PointSet* AP = &P->pointSets[ps_id];    // original points
    PointSet* AQ = &Q->pointSets[ps_id];    // mutated points    
    
    Point* p = &(AP->points[point_id]);    // original
    Point* q = &(AQ->points[point_id]);    // mutated point
    
    hiprandState localState = state[ps_id*N_POINTS + point_id];
    
    // no mutation
    if (mutate_pointset[ps_id] == 0 || cuda_randomChoice(1 - POINT_MUTATION_PROB, &localState)) {
        *q = *p;        
    }
    // mix
    else if (mutate_pointset[ps_id] == 1) {
        mixFlat(p, AP, q, obstacles, &localState);
    }
    // randomMove
    else {
        randomMoveFlat(p, q, obstacles, &localState);
    }
}

// Q = mutation of the X% best portion of P
// llegeix de P, escriu a Q
void mutate(Population** gpu_P, Population** gpu_Q) {
    
    tic(&mutationTime);

    ForEachGPU {
        hipSetDevice(GPU_X);
            
        int n_threads = 32; //sqrt(1024)
        int width = N / n_threads;
        int height = N_POINTS / n_threads;

        dim3 gridSize(width, height, 1);
        dim3 blockSize(n_threads, n_threads, 1);
        /*
        kernel_mutateFlat<<<gridSize, blockSize>>>(gpu_P[GPU_X], 
                                                   gpu_Q[GPU_X], 
                                                   gpu_obstacles[GPU_X], 
                                                   devStates[GPU_X]);
        */
        kernel_mutate<<<nBlocks, nThreads>>>(gpu_P[GPU_X], gpu_Q[GPU_X], 
                                             gpu_obstacles[GPU_X], devStates[GPU_X]);
        
        
        checkCudaError((char *) "kernel call in mutate");
    }
    ForEachGPU { 
        hipSetDevice(GPU_X);
        hipDeviceSynchronize(); 
    }
    
    toc(&mutationTime);
}

void dump(PointSet** C) {
    for (int i = 0; i < N_POINTS; ++i) {
        printf("%f %f %f\n", C[bestGPU]->points[i].x, C[bestGPU]->points[i].y, C[bestGPU]->points[i].z);
    }
}

__device__ void pork(PointSet* p1, PointSet* p2, PointSet* child, hiprandState* localState) {
    for (int i = 0; i < N_POINTS; ++i) {
        Point* p = &child->points[i];        
        if (cuda_randomChoice(0.5, localState)) {
            *p = p1->points[i];
        }
        else {
            *p = p2->points[i];
        }
    }
}

__global__ void kernel_reproduce(Population* P, Population* Q, int* idxs, 
                                                        hiprandState* state) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idxs[id] < N_SURVIVORS) {
        Q->pointSets[id] = P->pointSets[id];
    }
    else {
        hiprandState localState = state[id];
        int ip1 = idxs[(unsigned int) (CURAND*(N_SURVIVORS-1))];
        PointSet* p1 = &P->pointSets[ip1];
        int ip2 = idxs[(unsigned int) (CURAND*(N_SURVIVORS-1))];
        PointSet* p2 = &P->pointSets[ip2];        
        PointSet* child = &Q->pointSets[id];
        pork(p1, p2, child, &localState);
    }
}

void reproduce(Population** gpu_P, Population** gpu_Q, int** gpu_idxs) {
    tic(&reproductionTime);

    ForEachGPU {
        hipSetDevice(GPU_X);
        // kernel 
        kernel_reproduce<<<nBlocks, nThreads>>>(gpu_P[GPU_X], gpu_Q[GPU_X], 
                                                gpu_idxs[GPU_X], devStates[GPU_X]);
        checkCudaError((char *) "kernel call in mutate");
    }
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();        
    }
    toc(&reproductionTime);
}

void DUMPInitialParams() {
    printf("%i\n", N_OBSTACLES);
    for (int i = 0; i < N_OBSTACLES; ++i) {
        Obstacle o = obstacles[i];
        printf("%f %f %f %f\n", o.centre.x, o.centre.y, o.centre.z, o.radius); 
    }
    printf("%i %i\n", N_POINTS, ITERATION_LIMIT);
    
}

void initTimes() {
    initialGenTime = mutationTime = evaluationTime = sortingTime = reproductionTime = 0;
}

void printTimes() {
    printf("CUDA genetic algorithm has finished:\n");
    printf("    Init gen:     %f s.\n", (double)initialGenTime/1000000);
    printf("    Mutations:    %f s.\n", (double)mutationTime/1000000);
    printf("    Evaluations:  %f s.\n", (double)evaluationTime/1000000);
    printf("    Sorting:      %f s.\n", (double)sortingTime/1000000);
    printf("    Reproduction: %f s.\n", (double)reproductionTime/1000000);
    printf("    Total time:   %f s.\n", (double)totalTime/1000000);
}

void initObstacles() {    
    
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            for (int k = 0; k < 3; ++k) {
                Point origin;
                origin.x = 3*i;
                origin.y = 3*j;
                origin.z = 3*k;

                obstacles[i*9 + j*3 + k].centre = origin;
                obstacles[i*9 + j*3 + k].radius = 1.0;
            }
        }
    }
    
    gpu_obstacles = (Obstacle**) malloc(nGPUs*sizeof(Obstacle*));
    
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipMalloc(&gpu_obstacles[GPU_X], sizeof(Obstacle)*N_OBSTACLES);
        hipMemcpy(gpu_obstacles[GPU_X], obstacles, sizeof(Obstacle)*N_OBSTACLES, hipMemcpyHostToDevice);
        checkCudaError((char *) "host -> gpu obstacles");
    }

    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }
}

void initDestinationPoint() {
    
    destination.x = destination.y = destination.z = 0.0;   
    
    gpu_destination = (Point**) malloc(nGPUs*sizeof(Point*));
    
    ForEachGPU {
        hipSetDevice(GPU_X);
        // malloc
        hipMalloc(&gpu_destination[GPU_X], sizeof(Point));
        checkCudaError((char *) "hipMalloc in initDestinationPoint");
        
        // copy    
        hipMemcpy(gpu_destination[GPU_X], &destination, sizeof(Point), hipMemcpyHostToDevice);
        checkCudaError((char *) "host -> gpu in initDestinationPoint");
    }
    
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }
}

void randomSetup() {
    srand(SEED);    
    seeds = (int*) malloc(nGPUs*sizeof(int));
    ForEachGPU {
        seeds[GPU_X] = rand();
    }

    devStates = (hiprandState**) malloc(nGPUs*sizeof(hiprandState*));
    
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipMalloc((void **)&devStates[GPU_X], N * sizeof(hiprandState));
        setup_kernel<<<nBlocks, nThreads>>>(devStates[GPU_X], seeds[GPU_X]);
        checkCudaError((char *) "setup random kernel");    
        
    }
    ForEachGPU {
        hipSetDevice(GPU_X);
        hipDeviceSynchronize();
    }
}

void cudaGenetic() {
    
    if (MULTI_GPU)
        hipGetDeviceCount(&nGPUs);
    else 
        nGPUs = 1;
    

    randomSetup();    
    initObstacles(); 
    initDestinationPoint();
    
    if (!DUMP) printf("Using %i GPUs.\n", nGPUs);
    
    tic(&totalTime);
    
    Population** gpu_P = (Population**) malloc(nGPUs*sizeof(Population*));
    Population** gpu_Q = (Population**) malloc(nGPUs*sizeof(Population*));

    PointSet** bestPointSets = (PointSet**) malloc(nGPUs*sizeof(PointSet*));
    
    int** gpu_idxs = (int**) malloc(nGPUs*sizeof(int*));
    
    // Malloc
    ForEachGPU {    
        hipSetDevice(GPU_X);
        hipMalloc((void **) &gpu_P[GPU_X], sizeof(Population));
        checkCudaError((char *) "hipMalloc of P");
        
        hipMalloc((void **) &gpu_Q[GPU_X], sizeof(Population));
        checkCudaError((char *) "hipMalloc of Q");    
    
        hipMalloc((void **) &gpu_idxs[GPU_X], sizeof(int)*N);
        checkCudaError((char *) "hipMalloc of idxs"); 
        
        bestPointSets[GPU_X] = (PointSet*) malloc(sizeof(PointSet));   
        
        // Prepare CDP for the max depth 'MAX_DEPTH'.
        hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);
    }
    
    if (DUMP) DUMPInitialParams();
    else initTimes();
     
    generateInitialPopulation(gpu_P, gpu_idxs);
      
    int it = 0;
    while (true) {

        mutate(gpu_P, gpu_Q);
        
        evaluate(gpu_Q);
        
        sort(gpu_Q, gpu_idxs, bestPointSets);
        
        if (DUMP) dump(bestPointSets);
        else {            
            printf("It: %i/%i Score: %f -> %f Best GPU: %i\n\n", 
                   it, ITERATION_LIMIT, bestPointSets[bestGPU]->score, GOAL_SCORE, bestGPU);
        }
        
        if (it >= ITERATION_LIMIT)  
            break;

        //////////////////////////////////////////////////       // bestPointSet->score <= GOAL_SCORE)
        
        // reproduce replaces the worst candidates with combinations
        // of better ones. from Q to P, so the population ends up in P
        // prepared for the next iteration
        reproduce(gpu_Q, gpu_P, gpu_idxs);
        it++;
    }
    // The result is bestPointSet
    
    toc(&totalTime);
    
    if (!DUMP) printTimes();
}

int main(int argc, char** argv) {
    cudaGenetic();
    return 0;
}

