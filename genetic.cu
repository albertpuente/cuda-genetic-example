/*
Compile with: 

    gcc genetic.c -o genetic -O2 -lm -std=c99 

    -O2      Optimization
    -lm      Link to math lib
    -std=c99 Use of for(;;;) with declaration among other things

Usage (3D viewer):

    ./genetic > data && ./geneticViewer data
    
Usage (debug):

    ./genetic

Jan Mas Rovira
Andrés Mingorance López
Albert Puente Encinas
*/

#include <stdio.h>  // e.g. printf
#include <stdlib.h> // e.g. malloc, RAND_MAX, exit
#include <math.h>   // e.g. sin, abs
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define CURAND hiprand_uniform(&localState)

// Genetic algorithm parameters
#define N 2048
#define N_POINTS 128
#define ITERATION_LIMIT 50
#define GOAL_SCORE -1.0
#define POINT_SET_MUTATION_PROB 0.5
#define POINT_MUTATION_PROB 0.01
#define N_SURVIVORS N/4
#define POINT_RADIUS 0.25
#define OBSTACLE_RADIUS 2.0
#define MAX_DELTA 2
#define MAX_TRIES 1e3   // max amount of times we tries to find a position for a point

// Obstacles
#define CHECK_OBSTACLES true
#define CHECK_COLLISIONS true

// Deterministic algorithm (testing purposes)
#define SEED 27
# define RAND01 ((float)rand()/(float)(RAND_MAX))

// c++ style
#define bool int
#define true 1
#define false 0

// Timers
unsigned long long mutationTime;
unsigned long long reproductionTime;
unsigned long long sortingTime;
unsigned long long evaluationTime;
unsigned long long initialGenTime;
unsigned long long totalTime;

inline void tic(unsigned long long* time) {
    struct timeval t;
    gettimeofday(&t, NULL);
    *time = t.tv_sec*1000000 + t.tv_usec - *time;
}
#define toc tic
//inline void toc(unsigned long long* time) { tic(time); }

// Output toggles
bool DUMP;

typedef struct {
    float x, y, z; // Position
} Point;

typedef struct {
    Point points[N_POINTS];
    float score;
} PointSet;

typedef struct {
    PointSet pointSets[N];
    float maxScore;
} Population;

typedef struct {
    Point centre;
    float radius;
} Obstacle;

#define N_OBSTACLES 27
Obstacle obstacles[N_OBSTACLES];
Point destination;

// CUDA Variables
unsigned int nThreads = 1024;
unsigned int nBlocks = N/nThreads;  // N multiple de nThreads
    
// GPU Pointers
Obstacle* gpu_obstacles;
Point* gpu_destination;

__device__ inline bool cuda_randomChoice(float probability, hiprandState* localState) {
    if (hiprand_uniform(localState) <= probability) return true;
    else return false;    
}

void checkCudaError(char msg[]) {
    hipError_t error;
    error = hipGetLastError();
    if (error) {
        printf("Error: %s: %s\n", msg, hipGetErrorString(error));
        exit(1);
    }
}

__device__ inline float cuda_dist(Point* a, Point* b) {
    return sqrt(pow(a->x - b->x, 2)+pow(a->y - b->y, 2)+pow(a->z - b->z, 2));
}

__device__ bool cuda_collides(Point* p, PointSet* PS, int from, int to, Obstacle* obstacles) {
    if (CHECK_COLLISIONS)
        for (int i = from; i < to; ++i) {
            if (cuda_dist(p, &PS->points[i]) < POINT_RADIUS*2) {
                return true;
            }
        }
    if (CHECK_OBSTACLES)
        for (int i = 0; i < N_OBSTACLES; ++i) {
            Obstacle o = obstacles[i];
            if (cuda_dist(p, &o.centre) < POINT_RADIUS + o.radius) {
                return true;
            }
        }
    return false;
}

__global__ void kernel_generateInitialPopulation(Population* P, 
                    Obstacle* obstacles, hiprandState* state) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    hiprandState localState = state[id];
    
    float range = POINT_RADIUS * pow((float)N_POINTS, 1.0f/3.0f) * 10;    
    
    /*
    float r1 = curand_uniform(&localState)/(float)(RAND_MAX);
    float r2 = curand_uniform(&localState);///(float)(RAND_MAX);
    float r3 = curand_uniform(&localState);///(float)(RAND_MAX);
    
    printf("%f %f %f\n", r1, r2, r3);
    */
    
    for (int j = 0; j < N_POINTS; ++j) {
        PointSet* PS = &(P->pointSets[id]);
        Point* p = &(PS->points[j]); // p is passed to 'collides' via PS
        p->x = CURAND * range + 12.5;
        p->y = CURAND * range + 12.5;
        p->z = CURAND * range + 12.5;
        
        int tries = 0;
        while (tries < MAX_TRIES && cuda_collides(p, PS, 0, j, obstacles)) {
            p->x = CURAND * range + 12.5;
            p->y = CURAND * range + 12.5;
            p->z = CURAND * 5.0 + 12.5;
            ++tries;
        }
        if (tries == MAX_TRIES) {
            printf("Error during the generation of the initial population\n");
            //exit(1);
        }
    }
}

__global__ void setup_kernel(hiprandState *state) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init ( 1234, id, 0, &state[id] );
}

void generateInitialPopulation(Population* gpu_P) {
    tic(&initialGenTime);
    
    //RANDOM SETUP
    hiprandState *devStates;
    hipMalloc((void **)&devStates, N * sizeof(hiprandState));
    setup_kernel<<<nBlocks, nThreads>>>(devStates);
    checkCudaError((char *) "setup random kernel");    
    //RANDOM END
    
    // kernel 
    kernel_generateInitialPopulation<<<nBlocks, nThreads>>>(gpu_P, gpu_obstacles, devStates);
    checkCudaError((char *) "kernel call in generateInitialPopulation");
    
    // wait
    hipDeviceSynchronize();
    toc(&initialGenTime);
}

__device__ inline float heur_1(Point* P) {
    return fabs(P->y - 3.0*sin(P->x/2.0)) + fabs(P->z - 3.0*cos(P->x/2.0));
}


__device__ inline float heur_2(Point* P, Point* destination) {
    return cuda_dist(P, destination);
}

__global__ void kernel_evaluate(Population* P, Point* destination) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    PointSet* C = &P->pointSets[id];
    C->score = 0;
    for (int j = 0; j < N_POINTS; j++) {
        Point* E = &C->points[j];
        C->score += heur_2(E, destination);
    }
}

void evaluate(Population* gpu_P) {
    tic(&evaluationTime);
    
    // kernel 
    kernel_evaluate<<<nBlocks, nThreads>>>(gpu_P, gpu_destination);
    checkCudaError((char *) "kernel call in generateInitialPopulation");
    
    // wait
    hipDeviceSynchronize();
    toc(&evaluationTime);
}

//////////////////////////////////////////
//////////////  CUDA QUICK SORT
//////         

#define MAX_DEPTH       16
#define INSERTION_SORT  32

__device__ void swap_pointSets(PointSet* a, PointSet* b) {
    PointSet aux = *a;
    *a = *b;
    *b = aux;
}


// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
__device__ void selection_sort(Population* P, int left, int right ) {
    for (int i = left ; i <= right ; ++i) {
        float min_score = P->pointSets[i].score;
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i + 1 ; j <= right ; ++j) {
            float score_j = P->pointSets[j].score;
            if (score_j < min_score) {
                min_idx = j;
                min_score = score_j;
            }
        }

        // Swap the values.
        if (i != min_idx) {
            swap_pointSets(&P->pointSets[i], &P->pointSets[min_idx]);
        }
    }
}


__device__ inline PointSet* candidate(Population* p, int ix) {
    return &p->pointSets[ix];
}

__device__ inline void swapCandidates(Population* p, int i, int j) {
    swap_pointSets(candidate(p, i), candidate(p, j));
}

// Partition algorithm (from wikipedia)
//////////////////////////////////////////////////////////////////////////////
//  // lo is the index of the leftmost element of the subarray              //
//  // hi is the index of the rightmost element of the subarray (inclusive) //
//  partition(A, lo, hi)                                                    //
//     pivotIndex := choosePivot(A, lo, hi)                                 //
//     pivotValue := A[pivotIndex]                                          //
//     // put the chosen pivot at A[hi]                                     //
//     swap A[pivotIndex] and A[hi]                                         //
//     storeIndex := lo                                                     //
//     // Compare remaining array elements against pivotValue = A[hi]       //
//     for i from lo to hi−1, inclusive                                     //
//         if A[i] <= pivotValue                                            //
//             swap A[i] and A[storeIndex]                                  //
//             storeIndex := storeIndex + 1                                 //
//     swap A[storeIndex] and A[hi]  // Move pivot to its final place       //
//     return storeIndex                                                    //
//////////////////////////////////////////////////////////////////////////////

__device__ int partition(Population* P, int lo, int hi) {
    int pivIx = (lo + hi)/2;
    float pivScore = candidate(P, pivIx)->score;
    swapCandidates(P, hi, pivIx);
    int stIx = lo;
    for (int i = lo; i < hi; ++i) {
        if (candidate(P, i)->score <= pivScore) {
            swapCandidates(P, i, stIx);
            ++stIx;
        }
    }
    swapCandidates(P, stIx, hi);
    return stIx;
}

// quicksort algorithm (from wikipedia)
/////////////////////////////////////
// quicksort(A, lo, hi):           //
//     if lo < hi:                 //
//       p := partition(A, lo, hi) //
//         quicksort(A, lo, p - 1) //
//         quicksort(A, p + 1, hi) //
/////////////////////////////////////
__global__ void dynamic_quicksort_jan(Population* P, int left, int right, int depth) {
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT) {
        selection_sort(P, left, right);
        return;
    }
        int piv = partition(P, left, right);
        
        // left
        if (left < piv - 1) {
            hipStream_t ls;
            hipStreamCreateWithFlags(&ls, hipStreamNonBlocking);
            dynamic_quicksort_jan<<< 1, 1, 0, ls >>>(P, left, piv - 1, depth + 1);
            hipStreamDestroy(ls);
        }

        //right
        if (piv + 1 < right) { 
            hipStream_t rs;
            hipStreamCreateWithFlags(&rs, hipStreamNonBlocking);
            dynamic_quicksort_jan<<< 1, 1, 0, rs >>>(P, piv + 1, right, depth + 1);
            hipStreamDestroy(rs);
        }    
}

__global__ void dynamic_quicksort(Population* P, int left, int right, int depth) {
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT) {
        selection_sort(P, left, right);
        return;
    }
    
    int lindex = left;
    int rindex = right;
    float pscore = P->pointSets[(left+right)/2].score; // Pivot

    // Do the partitioning.
    while (lindex <= rindex) {
        // Find the next left- and right-hand values to swap
        float lscore = P->pointSets[lindex].score; 
        float rscore = P->pointSets[rindex].score;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        // TODO: dicotomic search
        while (lscore < pscore) {
            lindex++;
            lscore = P->pointSets[lindex].score; 
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        // TODO: dicotomic search
        while (rscore > pscore) {
            rindex--;
            rscore = P->pointSets[rindex].score;
        }

        // If the swap points are valid, do the swap!
        if (lindex <= rindex) {
            
            // TODO: This needs to be improved, we can sort a vector
            // of indices instead of copying the whole pointSets.
            swap_pointSets(&P->pointSets[lindex], &P->pointSets[rindex]);
            
            lindex++;
            rindex--;
        }
    }

    // Now the recursive part
    // Launch a new block to sort the left part.
    if (left < rindex) {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        dynamic_quicksort<<< 1, 1, 0, s >>>(P, left, rindex, depth + 1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if (lindex < right) {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        dynamic_quicksort<<< 1, 1, 0, s1 >>>(P, lindex, right, depth + 1);
        hipStreamDestroy(s1);
    }
}


void sort(Population* gpu_P) {
    tic(&sortingTime);
    
    // Prepare CDP for the max depth 'MAX_DEPTH'.
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);
    
    dynamic_quicksort<<<1, 1>>>(gpu_P, 0, N-1, 0);
    checkCudaError((char *) "kernel call in mutate");
    hipDeviceSynchronize();
    toc(&sortingTime);
}

__device__ void mix(PointSet* AP, PointSet* AQ, Obstacle* obstacles, 
                                                    hiprandState* localState) {

    for (int i = 0; i < N_POINTS; ++i) {
        
        if (!cuda_randomChoice(POINT_MUTATION_PROB, localState)) {
            AQ->points[i] = AP->points[i];
            continue;
        }           
   
        int tries = 0;
        Point p;
        while (tries < MAX_TRIES) {            
            // Choose a reference point
            int j = hiprand_uniform(localState)*(N_POINTS-1);
            
            // Calculate the direction from AP[i] to AP[j]
            float dx =  AP->points[j].x - AP->points[i].x;
            float dy =  AP->points[j].y - AP->points[i].y;
            float dz =  AP->points[j].z - AP->points[i].z;
            // "Normalization" ||direction|| = 0.5
            float norm = sqrt(pow(dx,2)+pow(dy,2)+pow(dz,2));
            norm *= (1.0/MAX_DELTA);
            norm /= hiprand_uniform(localState); // move a random portion of MAX_DELTA
            if (norm < 1e-4 && norm > -1e-4) {
                dx = 0;
                dy = 0;
                dz = 0;
            }
            else {
                dx /= norm;
                dy /= norm;  
                dz /= norm;   
            }       
            
            // 50% of getting closer, 50% of getting further away from the ref point
            if (cuda_randomChoice(0.5f, localState)) {
                p.x = AP->points[i].x + dx;
                p.y = AP->points[i].y + dy;
                p.z = AP->points[i].z + dz;
            }
            else {
                p.x = AP->points[i].x - dx;
                p.y = AP->points[i].y - dy;
                p.z = AP->points[i].z - dz;
            }
            // if the point doesn't collide with a point that has already moved
            if (!cuda_collides(&p, AQ, 0, i, obstacles) &&
                // and it doesn't collide with a point that has yet to be moved
                // (this 2nd check prevents inconsistencies like a point being unable to move at all)
                !cuda_collides(&p, AP, i + 1, N_POINTS, obstacles))
                    break;
            ++tries;
        }
        if (tries == MAX_TRIES) {
            //printf("Error during the mix() of points\n");
            //exit(1);
            p = AP->points[i];
        }
        AQ->points[i] = p;
    }
}

__device__ void randomMove(PointSet* AP, PointSet* AQ, Obstacle* obstacles, 
                                                hiprandState* localState) {
    for (int i = 0; i < N_POINTS; ++i) {
        
        if (!cuda_randomChoice(POINT_MUTATION_PROB, localState)) {
            AQ->points[i] = AP->points[i];
            continue;
        }
        int tries = 0;
        Point p;
        while (tries < MAX_TRIES) {
            p.x = AP->points[i].x + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
            p.y = AP->points[i].y + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
            p.z = AP->points[i].z + (hiprand_uniform(localState)-0.5)*2*MAX_DELTA;
            // if the point doesn't collide with a point that has already moved
            if (!cuda_collides(&p, AQ, 0, i, obstacles) &&
                // and it doesn't collide with a point that has yet to be moved
                // (this 2nd check prevents inconsistencies like a point being unable to move at all)
                !cuda_collides(&p, AP, i + 1, N_POINTS, obstacles))
                    break;
            ++tries;
        }
        if (tries == MAX_TRIES) {
            //printf("Error during the mix() of points\n");
            //exit(1);
            p = AP->points[i];
        }

        AQ->points[i] = p;
    } 
}

__global__ void kernel_mutate(Population* P, Population* Q, Obstacle* obstacles,
                                    hiprandState* state) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    hiprandState localState = state[id];
    
    PointSet* AP = &P->pointSets[id];    // original points
    PointSet* AQ = &Q->pointSets[id];    // mutated points
    if (cuda_randomChoice(POINT_SET_MUTATION_PROB, &localState)) { // Mutate
        if (cuda_randomChoice(0.5f, &localState)) {
            mix(AP, AQ, obstacles, &localState);
        }
        else {
            randomMove(AP, AQ, obstacles, &localState);
        }
    }        
    else { // Copy
        *AQ = *AP;
    }
}

// Q = mutation of the X% best portion of P
// llegeix de P, escriu a Q
void mutate(Population* gpu_P, Population* gpu_Q) {
    tic(&mutationTime);
    
     //RANDOM SETUP
    hiprandState *devStates;
    hipMalloc((void **)&devStates, N * sizeof(hiprandState));
    setup_kernel<<<nBlocks, nThreads>>>(devStates);
    checkCudaError((char *) "setup random kernel");    
    //RANDOM END
    
    // kernel 
    kernel_mutate<<<nBlocks, nThreads>>>(gpu_P, gpu_Q, gpu_obstacles, devStates);
    checkCudaError((char *) "kernel call in mutate");
    hipDeviceSynchronize();
    
    toc(&mutationTime);
}

void dump(PointSet* C) {
    for (int i = 0; i < N_POINTS; ++i) {
        printf("%f %f %f\n", C->points[i].x, C->points[i].y, C->points[i].z);
    }
}

__device__ void pork(PointSet* p1, PointSet* p2, PointSet* child, hiprandState* localState) {
    for (int i = 0; i < N_POINTS; ++i) {
        Point* p = &child->points[i];        
        if (cuda_randomChoice(0.5, localState)) {
            *p = p1->points[i];
        }
        else {
            *p = p2->points[i];
        }
    }
}

__global__ void kernel_reproduce(Population* P, Population* Q, hiprandState* state) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id < N_SURVIVORS) {
        Q->pointSets[id] = P->pointSets[id];
    }
    else {
        hiprandState localState = state[id];
        PointSet* p1 = &P->pointSets[(unsigned int) (CURAND*(N_SURVIVORS-1))];
        PointSet* p2 = &P->pointSets[(unsigned int) (CURAND*(N_SURVIVORS-1))];        
        PointSet* child = &Q->pointSets[id];
        pork(p1, p2, child, &localState);
    }
}

void reproduce(Population* gpu_P, Population* gpu_Q) {
    tic(&reproductionTime);
    
    //RANDOM SETUP
    hiprandState *devStates;
    hipMalloc((void **)&devStates, N * sizeof(hiprandState));
    setup_kernel<<<nBlocks, nThreads>>>(devStates);
    checkCudaError((char *) "setup random kernel");    
    //RANDOM END
    
    // kernel 
    kernel_reproduce<<<nBlocks, nThreads>>>(gpu_P, gpu_Q, devStates);
    checkCudaError((char *) "kernel call in mutate");   
    hipDeviceSynchronize();
    
    toc(&reproductionTime);
}

void getBestFromGPU(Population* gpu_P, PointSet* best) {
    
    // ?????????????????????????????????????????????????????????????? 
    hipMemcpy(best, gpu_P, sizeof(PointSet), hipMemcpyDeviceToHost);
    checkCudaError((char *) "gpu -> host getBestFromGPU");    
    hipDeviceSynchronize();
}

void DUMPInitialParams() {
    printf("%i\n", N_OBSTACLES);
    for (int i = 0; i < N_OBSTACLES; ++i) {
        Obstacle o = obstacles[i];
        printf("%f %f %f %f\n", o.centre.x, o.centre.y, o.centre.z, o.radius); 
    }
    printf("%i %i\n", N_POINTS, ITERATION_LIMIT);
    
}

void initTimes() {
    initialGenTime = mutationTime = evaluationTime = sortingTime = reproductionTime = 0;
}

void printTimes() {
    printf("CUDA genetic algorithm has finished:\n");
    printf("    Init gen:     %f s.\n", (double)initialGenTime/1000000);
    printf("    Mutations:    %f s.\n", (double)mutationTime/1000000);
    printf("    Evaluations:  %f s.\n", (double)evaluationTime/1000000);
    printf("    Sorting:      %f s.\n", (double)sortingTime/1000000);
    printf("    Reproduction: %f s.\n", (double)reproductionTime/1000000);
    printf("    Total time:   %f s.\n", (double)totalTime/1000000);
}

void initObstacles() {    
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            for (int k = 0; k < 3; ++k) {
                Point origin;
                origin.x = 3*i;
                origin.y = 3*j;
                origin.z = 3*k;

                obstacles[i*9 + j*3 + k].centre = origin;
                obstacles[i*9 + j*3 + k].radius = 1.0;
            }
        }
    }
    
    hipMalloc(&gpu_obstacles, sizeof(Obstacle)*N_OBSTACLES);
    hipMemcpy(gpu_obstacles, obstacles, sizeof(Obstacle)*N_OBSTACLES, hipMemcpyHostToDevice);
    checkCudaError((char *) "host -> gpu obstacles");
    
    hipDeviceSynchronize();
}

void initDestinationPoint() {
    
    destination.x = destination.y = destination.z = 0.0;   
    
    // malloc
    hipMalloc(&gpu_destination, sizeof(Point));
    checkCudaError((char *) "hipMalloc in initDestinationPoint");
    
    // copy    
    hipMemcpy(gpu_destination, &destination, sizeof(Point), hipMemcpyHostToDevice);
    checkCudaError((char *) "host -> gpu in initDestinationPoint");
}

void cudaGenetic() {
    srand(SEED);  
    
    initObstacles();    
    initDestinationPoint();
    
    tic(&totalTime);   
    
    // Malloc
    Population* gpu_P;
    Population* gpu_Q;
    hipMalloc((void **) &gpu_P, sizeof(Population));
    checkCudaError((char *) "hipMalloc of P");
    hipMalloc((void **) &gpu_Q, sizeof(Population));
    checkCudaError((char *) "hipMalloc of Q");    
    
    PointSet* bestPointSet = (PointSet*) malloc(sizeof(PointSet));

    if (DUMP) DUMPInitialParams();
    else initTimes();
    
    generateInitialPopulation(gpu_P);
    
    int it = 0;
    while (true) {
        mutate(gpu_P, gpu_Q);
        evaluate(gpu_Q);
        sort(gpu_Q);        
        getBestFromGPU(gpu_Q, bestPointSet);
        
        if (DUMP) dump(bestPointSet);
        else {            
            printf("\nIt: %i/%i Score: %f -> %f\n", 
                   it, ITERATION_LIMIT, bestPointSet->score, GOAL_SCORE);
        }
        
        if (it >= ITERATION_LIMIT || bestPointSet->score <= GOAL_SCORE) 
            break;
        
        // reproduce replaces the worst candidates with combinations
        // of better ones. from Q to P, so the population ends up in P
        // prepared for the next iteration
        reproduce(gpu_Q, gpu_P);
        it++;
    }
    // The result is bestPointSet
    
    toc(&totalTime);
    
    if (!DUMP) printTimes();
}

int main(int argc, char** argv) {
    DUMP = (argc == 1);
    cudaGenetic();
    return 0;
}

